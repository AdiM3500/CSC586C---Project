
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include<iterator>
#include <cmath>
#include <optional>
#include <utility>
#include <vector>
#include <array>
#include <variant>
#include <queue>
#include <tuple>
#include <chrono>
#include <random>
#include <immintrin.h>
#include <emmintrin.h>
#include <omp.h>


//global variable to keep track of the number of joins that occured
long long int joins_count = 0;
long long int naive_joins_count = 0;
long long int naive_naive_joins_count = 0;
long long int tiling_joins_count = 0;
using coordinate_t = std::pair <int, int>;
using pair_of_coordinates_t = std::pair<coordinate_t, coordinate_t>;
using rectangle_list_t = std::vector<pair_of_coordinates_t>;
using spatial_join_list_t = std::vector<std::pair < pair_of_coordinates_t, pair_of_coordinates_t> >;

class RectangleSet_Coordinates {

public:

	int x_min[100000];
	int y_min[100000];
	int x_max[100000];
	int y_max[100000];
	int count = 0;

	RectangleSet_Coordinates(rectangle_list_t const& coords_to_insert) {

		//x_min.resize(coords_to_insert.size()); y_min.resize(coords_to_insert.size()); x_max.resize(coords_to_insert.size()); y_max.resize(coords_to_insert.size());

		for (auto i = 0; i < coords_to_insert.size(); i++) {

			x_min[i] = coords_to_insert[i].first.first;
			y_min[i] = coords_to_insert[i].first.second;

			x_max[i] = coords_to_insert[i].second.first;
			y_max[i] = coords_to_insert[i].second.second;
			count++;
		}

	}


};



void NAIVE_SOA_SPATIAL_JOIN(RectangleSet_Coordinates R, RectangleSet_Coordinates S) {

	for (auto i = 0; i < R.count; i++) {

		//inner loop iterating through S
		for (auto j = 0; j < S.count; j++) {

			if ((R.x_min[i] < S.x_max[j])
				&& (R.x_max[i] > S.x_min[j])
				&& (R.y_min[i] < S.y_max[j])
				&& (R.y_max[i] > S.y_min[j])
				) {

				naive_joins_count += 1;
			}


			//condition to check if a particular MBR object in R overlaps with ANY objects in S

		}
	}

}


void TILING_SOA_SPATIAL_JOIN(RectangleSet_Coordinates R, RectangleSet_Coordinates S) {

	//outer loop iterating through R
	for (auto i = 0; i < R.count - 7; i += 8) {

		for (auto j = 0; j < S.count; j++) {


				for (auto k = 0; k < 8; k++) {

					if ((R.x_min[i + k] < S.x_max[j])
						&& (R.x_max[i + k] > S.x_min[j])
						&& (R.y_min[i + k] < S.y_max[j])
						&& (R.y_max[i + k] > S.y_min[j])
						) {

						tiling_joins_count += 1;
					}

				}
			
		}

	}


}


void SIMD_SPATIAL_JOIN(RectangleSet_Coordinates R, RectangleSet_Coordinates S) {

//	int  num_threads = 6;
//	omp_set_num_threads(num_threads);

	for (auto i = 0; i < R.count - 7; i += 8) {

		#pragma omp parallel for reduction (+:joins_count) 
		for (auto j = 0; j < S.count; j++) {

            for(auto k = 0; k < 8; k+=4){

                    

				__m128i setR_min_x = _mm_set_epi32(R.x_min[i + k], R.x_min[i + k + 1], R.x_min[i + k + 2], R.x_min[i + k + 3]);
				__m128i setR_min_y = _mm_set_epi32(R.y_min[i + k], R.y_min[i + k + 1], R.y_min[i + k + 2], R.y_min[i + k + 3]);
				__m128i	setR_max_x = _mm_set_epi32(R.x_max[i + k], R.x_max[i+ k + 1 ], R.x_max[i + k + 2], R.x_max[i + k + 3]);
				__m128i	setR_max_y = _mm_set_epi32(R.y_max[i + k], R.y_max[i + k + 1], R.y_max[i + k + 2], R.y_max[i + k + 3]);


				__m128i setS_min_x = _mm_set_epi32(S.x_min[j], S.x_min[j], S.x_min[j], S.x_min[j]);
				__m128i setS_min_y = _mm_set_epi32(S.y_min[j], S.y_min[j], S.y_min[j], S.y_min[j]);
				__m128i	setS_max_x = _mm_set_epi32(S.x_max[j], S.x_max[j], S.x_max[j], S.x_max[j]);
				__m128i	setS_max_y = _mm_set_epi32(S.y_max[j], S.y_max[j], S.y_max[j], S.y_max[j]);

				auto first_comparison = _mm_cmplt_epi32(setR_min_x, setS_max_x);
				auto second_comparison = _mm_cmpgt_epi32(setR_max_x, setS_min_x);
				auto third_comparison = _mm_cmplt_epi32(setR_min_y, setS_max_y);
				auto fourth_comparison = _mm_cmpgt_epi32(setR_max_y, setS_min_y);

               

				joins_count += _mm_extract_epi32(first_comparison, 3) * _mm_extract_epi32(second_comparison, 3) * _mm_extract_epi32(third_comparison, 3) * _mm_extract_epi32(fourth_comparison, 3);
				joins_count += _mm_extract_epi32(first_comparison, 2) * _mm_extract_epi32(second_comparison, 2) * _mm_extract_epi32(third_comparison, 2) * _mm_extract_epi32(fourth_comparison, 2);
				joins_count += _mm_extract_epi32(first_comparison, 1) * _mm_extract_epi32(second_comparison, 1) * _mm_extract_epi32(third_comparison, 1) * _mm_extract_epi32(fourth_comparison, 1);
				joins_count += _mm_extract_epi32(first_comparison, 0) * _mm_extract_epi32(second_comparison, 0) * _mm_extract_epi32(third_comparison, 0) * _mm_extract_epi32(fourth_comparison, 0);
		

            }


			//8 Rectangles of set R disintegrated into its smallest elements i.e. coordinates, which are 32 bit integers. Packed into an _m128i datatype. Each _m128i datatype can store 4 32-bit integers

			/* 
				__m128i setR_min_x = _mm_set_epi32(R.x_min[i], R.x_min[i + 1], R.x_min[i + 2], R.x_min[i + 3]);
				__m128i setR_min_y = _mm_set_epi32(R.y_min[i], R.y_min[i + 1], R.y_min[i + 2], R.y_min[i + 3]);
				__m128i	setR_max_x = _mm_set_epi32(R.x_max[i], R.x_max[i+ 1 ], R.x_max[i + 2], R.x_max[i + 3]);
				__m128i	setR_max_y = _mm_set_epi32(R.y_max[i], R.y_max[i + 1], R.y_max[i + 2], R.y_max[i + 3]);


				__m128i setS_min_x = _mm_set_epi32(S.x_min[j], S.x_min[j], S.x_min[j], S.x_min[j]);
				__m128i setS_min_y = _mm_set_epi32(S.y_min[j], S.y_min[j], S.y_min[j], S.y_min[j]);
				__m128i	setS_max_x = _mm_set_epi32(S.x_max[j], S.x_max[j], S.x_max[j], S.x_max[j]);
				__m128i	setS_max_y = _mm_set_epi32(S.y_max[j], S.y_max[j], S.y_max[j], S.y_max[j]);


				auto first_comparison = _mm_cmplt_epi32(setR_min_x, setS_max_x);
				auto second_comparison = _mm_cmpgt_epi32(setR_max_x, setS_min_x);
				auto third_comparison = _mm_cmplt_epi32(setR_min_y, setS_max_y);
				auto fourth_comparison = _mm_cmpgt_epi32(setR_max_y, setS_min_y);


				joins_count += _mm_extract_epi32(first_comparison, 3) * _mm_extract_epi32(second_comparison, 3) * _mm_extract_epi32(third_comparison, 3) * _mm_extract_epi32(fourth_comparison, 3);
				joins_count += _mm_extract_epi32(first_comparison, 2) * _mm_extract_epi32(second_comparison, 2) * _mm_extract_epi32(third_comparison, 2) * _mm_extract_epi32(fourth_comparison, 2);
				joins_count += _mm_extract_epi32(first_comparison, 1) * _mm_extract_epi32(second_comparison, 1) * _mm_extract_epi32(third_comparison, 1) * _mm_extract_epi32(fourth_comparison, 1);
				joins_count += _mm_extract_epi32(first_comparison, 0) * _mm_extract_epi32(second_comparison, 0) * _mm_extract_epi32(third_comparison, 0) * _mm_extract_epi32(fourth_comparison, 0);

				
				// 4 more now:
				 setR_min_x = _mm_set_epi32(R.x_min[i + 4], R.x_min[i + 5], R.x_min[i + 6], R.x_min[i + 7]);
				 setR_min_y = _mm_set_epi32(R.y_min[i + 4], R.y_min[i + 5], R.y_min[i + 6], R.y_min[i + 7]);
				 setR_max_x = _mm_set_epi32(R.x_max[i + 4], R.x_max[i + 5], R.x_max[i + 6], R.x_max[i + 7]);
				 setR_max_y = _mm_set_epi32(R.y_max[i + 4], R.y_max[i + 5], R.y_max[i + 6], R.y_max[i + 7]);


			//	 setS_min_x = _mm_set_epi32(S.x_min[j], S.x_min[j], S.x_min[j], S.x_min[j]);
			//	 setS_min_y = _mm_set_epi32(S.y_min[j], S.y_min[j], S.y_min[j], S.y_min[j]);
			 //    setS_max_x = _mm_set_epi32(S.x_max[j], S.x_max[j], S.x_max[j], S.x_max[j]);
			//	 setS_max_y = _mm_set_epi32(S.y_max[j], S.y_max[j], S.y_max[j], S.y_max[j]);


				 first_comparison = _mm_cmplt_epi32(setR_min_x, setS_max_x);
				 second_comparison = _mm_cmpgt_epi32(setR_max_x, setS_min_x);
				 third_comparison = _mm_cmplt_epi32(setR_min_y, setS_max_y);
				 fourth_comparison = _mm_cmpgt_epi32(setR_max_y, setS_min_y);


				joins_count += _mm_extract_epi32(first_comparison, 3) * _mm_extract_epi32(second_comparison, 3) * _mm_extract_epi32(third_comparison, 3) * _mm_extract_epi32(fourth_comparison, 3);
				joins_count += _mm_extract_epi32(first_comparison, 2) * _mm_extract_epi32(second_comparison, 2) * _mm_extract_epi32(third_comparison, 2) * _mm_extract_epi32(fourth_comparison, 2);
				joins_count += _mm_extract_epi32(first_comparison, 1) * _mm_extract_epi32(second_comparison, 1) * _mm_extract_epi32(third_comparison, 1) * _mm_extract_epi32(fourth_comparison, 1);
				joins_count += _mm_extract_epi32(first_comparison, 0) * _mm_extract_epi32(second_comparison, 0) * _mm_extract_epi32(third_comparison, 0) * _mm_extract_epi32(fourth_comparison, 0);


			*/

			
		}

	}
}
			

class RectangleSet {

public:

	//list of mbrsspa
	rectangle_list_t mbrs;

	//spatial join function that returns a set of rectangles that overlap with each other
	spatial_join_list_t spatial_join(rectangle_list_t R, rectangle_list_t S);

	//naive spatial join function
	spatial_join_list_t naive_spatial_join(rectangle_list_t R, rectangle_list_t S);

	spatial_join_list_t tiling_spatial_join(rectangle_list_t R, rectangle_list_t S);

	RectangleSet(rectangle_list_t const& mbrs_to_insert) {

		mbrs.resize(mbrs_to_insert.size());
		std::copy(mbrs_to_insert.cbegin(), mbrs_to_insert.cend(), mbrs.begin());

	}

	~RectangleSet() {

		std::cout << "object destroyed" << std::endl;
	}
};


spatial_join_list_t RectangleSet::naive_spatial_join(rectangle_list_t R, rectangle_list_t S) {

	//R x S
	spatial_join_list_t spatial_list;
	spatial_list.reserve(10000000);
	//outer loop iterating through R
	for (auto i = 0; i < R.size(); i++) {
		//inner loop iterating through S
		for (auto j = 0; j < S.size(); j++) {

			//condition to check if a particular MBR object in R overlaps with ANY objects in S
			if ((R[i].first.first < S[j].second.first)
				&& (S[j].first.first < R[i].second.first)
				&& (R[i].first.second < S[j].second.second)
				&& (S[j].first.second < R[i].second.second)
				) {

				//spatial_list.push_back({ R[i], S[j] });
				naive_naive_joins_count += 1;


			}



		}
	}


	return spatial_list;
}

//Function inspired from https://stackoverflow.com/questions/7560114/random-number-c-in-some-range


rectangle_list_t random_rectangle_generator() {

	rectangle_list_t random_rectangle_list;
	std::random_device rd; // obtain a random number from hardware
	std::mt19937 gen(rd()); // seed the generator
	std::uniform_int_distribution<> distr(1, 1000); // define the range

	//auto const length = 100000u;
	//random_rectangle_list.reserve(length);

	for (auto n = 0; n < 100000; n++) {

		int min_x = distr(gen);
		int min_y = distr(gen);
		int max_x, max_y, length, width;


		do {

			max_x = distr(gen) + min_x;
			max_y = distr(gen) + min_y;
			length = max_x - min_x;
			width = max_y - min_y;
		} while ((length >= 20) || (width >= 10));




		random_rectangle_list.push_back({ {min_x,min_y}, {max_x,max_y} });
	}

	return random_rectangle_list;

}


rectangle_list_t random_rectangle_generator_soa() {

	rectangle_list_t random_rectangle_list;

	std::random_device rd; // obtain a random number from hardware
	std::mt19937 gen(rd()); // seed the generator
	std::uniform_int_distribution<> distr(1, 1000); // define the range

	for (auto n = 0; n < 100000; n++) {

		int min_x = distr(gen);
		int min_y = distr(gen);
		int max_x, max_y, length, width;


		do {

			max_x = distr(gen) + min_x;
			max_y = distr(gen) + min_y;
			length = max_x - min_x;
			width = max_y - min_y;
		} while ((length >= 20) || (width >= 10));




		random_rectangle_list.push_back({ {min_x,min_y}, {max_x,max_y} });
	}

	return random_rectangle_list;

}


//cuda kernel

__global__
void cuda_join(int* dev_min_x1, int* dev_max_x2, int* dev_max_x1, int* dev_min_x2, int* dev_min_y1, int* dev_max_y2, 
				int* dev_max_y1, int* dev_min_y2, int n, int n_phase2, int* dev_block_reduction_array){

__shared__ int pract_a[1024];

	    int th_id = blockIdx.x * blockDim.x + threadIdx.x;

	pract_a[threadIdx.x] = 0;

	for(int i = 0; i < (n-7); i+=8) {
							

		for(int j = ( th_id * ceil(n/25000) ); j < ( (th_id + 1) * ceil(n/25000) ); j++) {
								
			for(int k = 0; k < 8; k++) {
				

       			 	pract_a[threadIdx.x] += ( (dev_min_x1[i + k] < dev_max_x2[j]) && (dev_max_x1[i + k] > dev_min_x2[j])
					&& (dev_min_y1[i + k] < dev_max_y2[j]) && (dev_max_y1[i + k] > dev_min_y2[j]) ) ;
						
     			

			}
	
    	}
	}

__syncthreads();
//phase 2:

	while( n_phase2 > 0 )
	{
		n_phase2 = ceil(n_phase2 / 2);
		if( threadIdx.x < n_phase2 )
		{
			pract_a[ threadIdx.x ] += pract_a[ threadIdx.x + n_phase2 ];

		}
	}

    __syncthreads();
    if( threadIdx.x == 0 )
	{
	
                dev_block_reduction_array[blockIdx.x] = pract_a[0];

          //  printf("%d on block %d: \n", dev_block_reduction_array[blockIdx.x], blockIdx.x);

	}

}

int main() {

	auto soa_setR = random_rectangle_generator_soa();
	auto soa_setS = random_rectangle_generator_soa();


	RectangleSet_Coordinates soa_R(soa_setR);
	RectangleSet_Coordinates soa_S(soa_setS);

	RectangleSet R(soa_setR);
	RectangleSet S(soa_setS);
	
	std::cout << "Rectangles generated. Starting Spatial Join function Benchmarking now------------------" << std::endl;
/*
	//**********NAIVE JOIN FUNCTION BENCHMARKING**********************
	//std::cout << std::endl;

	//spatial_join_list_t OLD_JOIN;

	//CLOCK START
	auto naive_start = std::chrono::high_resolution_clock::now();

	//CALL THE FUNCTION
	NAIVE_SOA_SPATIAL_JOIN(soa_R, soa_S);

	//CLOCK END
	auto naive_end = std::chrono::high_resolution_clock::now();

	auto naive_elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>
		(naive_end - naive_start);

	std::cout << "Time in microseconds (naive SoA approach): " << naive_elapsed_time.count() << "us" << std::endl;


	std::cout << "Number of joins (naive SoA approach): " << naive_joins_count << std::endl << std::endl;

*/

	

	//**********TILING JOIN FUNCTION BENCHMARKING**********************
	std::cout << std::endl;


	//CLOCK START
	auto mid_start = std::chrono::high_resolution_clock::now();

	//CALL THE FUNCTION
	TILING_SOA_SPATIAL_JOIN(soa_R, soa_S);

	//CLOCK END
	auto mid_end = std::chrono::high_resolution_clock::now();

	auto mid_elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>
		(mid_end - mid_start);

	std::cout << "Time in microseconds (Tiling approach): " << mid_elapsed_time.count() << "us" << std::endl;


	std::cout << "Number of joins (Tiling approach): " << tiling_joins_count << std::endl << std::endl;




/*
	
	//**********NAIVE_NAIVE JOIN FUNCTION BENCHMARKING**********************
	std::cout << std::endl;

	spatial_join_list_t OLD_JOIN;

	//CLOCK START
	auto naive_naive_start = std::chrono::high_resolution_clock::now();

	//CALL THE FUNCTION
	OLD_JOIN = R.naive_spatial_join(R.mbrs, S.mbrs);

	//CLOCK END
	auto naive_naive_end = std::chrono::high_resolution_clock::now();

	auto naive_naive_elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>
		(naive_naive_end - naive_naive_start);

	std::cout << "Time in microseconds (naive approach): " << naive_naive_elapsed_time.count() << "us" << std::endl;


	std::cout << "Number of joins (naive approach): " << naive_naive_joins_count << std::endl << std::endl;

*/
	

	// **********OPTIMIZED SIMD JOIN FUNCTION BENCHMARKING**************
	std::cout << std::endl;

	//spatial_join_list_t JOIN;
	//CLOCK START
	auto start = std::chrono::high_resolution_clock::now();

	//CALL THE FUNCTION
	SIMD_SPATIAL_JOIN(soa_R, soa_S);

	//CLOCK END
	auto end = std::chrono::high_resolution_clock::now();

	auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>
		(end - start);
	std::cout << "Time in microseconds (optimized SIMD approach): " << elapsed_time.count() << "us" << std::endl;

	std::cout << "Number of joins (optimized SIMD approach): " << joins_count << std::endl;



//CUDA CODE
int n = 100000; //size of array

int host_block_reduction_array[25];

for(int i = 0; i < 25; i++){


    host_block_reduction_array[i] = 0;
}

int* dev_block_reduction_array;



hipMalloc((void**)&dev_block_reduction_array, sizeof(int)*25);
    hipMemcpy(dev_block_reduction_array, host_block_reduction_array, sizeof(int)*25, hipMemcpyHostToDevice);

int* dev_min_x1;
int* dev_max_x2;

int* dev_max_x1;
int* dev_min_x2;

int* dev_min_y1;
int* dev_max_y2;

int* dev_max_y1;
int* dev_min_y2;


hipMalloc((void**)&dev_min_x1, sizeof(int)*n );
hipMemcpy(dev_min_x1, soa_R.x_min, sizeof(int)*n, hipMemcpyHostToDevice);
hipMalloc((void**)&dev_max_x2, sizeof(int)*n );
hipMemcpy(dev_max_x2, soa_S.x_max, sizeof(int)*n, hipMemcpyHostToDevice);

hipMalloc((void**)&dev_max_x1, sizeof(int)*n );
hipMemcpy(dev_max_x1, soa_R.x_max, sizeof(int)*n, hipMemcpyHostToDevice);
hipMalloc((void**)&dev_min_x2, sizeof(int)*n );
hipMemcpy(dev_min_x2, soa_S.x_min, sizeof(int)*n, hipMemcpyHostToDevice);

hipMalloc((void**)&dev_min_y1, sizeof(int)*n );
hipMemcpy(dev_min_y1, soa_R.y_min, sizeof(int)*n, hipMemcpyHostToDevice);
hipMalloc((void**)&dev_max_y2, sizeof(int)*n );
hipMemcpy(dev_max_y2, soa_S.y_max, sizeof(int)*n, hipMemcpyHostToDevice);

hipMalloc((void**)&dev_max_y1, sizeof(int)*n );
hipMemcpy(dev_max_y1, soa_R.y_max, sizeof(int)*n, hipMemcpyHostToDevice);
hipMalloc((void**)&dev_min_y2, sizeof(int)*n );
hipMemcpy(dev_min_y2, soa_S.y_min, sizeof(int)*n, hipMemcpyHostToDevice);

//CLOCK START
	auto cuda_start = std::chrono::high_resolution_clock::now();

	//CALL THE FUNCTION
cuda_join<<<25,1000>>> (dev_min_x1, dev_max_x2, dev_max_x1, dev_min_x2, dev_min_y1, dev_max_y2, dev_max_y1, 
                        dev_min_y2, n, 1024, dev_block_reduction_array);

hipDeviceSynchronize();
//CLOCK END
	auto cuda_end = std::chrono::high_resolution_clock::now();

	auto cuda_elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>
		(cuda_end - cuda_start);
	std::cout << "Time in microseconds (optimized CUDA approach): " << cuda_elapsed_time.count() << "us" << std::endl;
hipMemcpy(host_block_reduction_array, dev_block_reduction_array, sizeof(int)*25, hipMemcpyDeviceToHost);

int total_block_reduction_counter = 0;
for(int x = 0; x < 25; x++){
   total_block_reduction_counter += host_block_reduction_array[x];
}
	std::cout << "Number of joins (optimized CUDA approach): " << (total_block_reduction_counter) <<std::endl;

}


