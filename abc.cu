#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>


__global__
void abc(int *count, int n) {
   __shared__ int a[256];

   int a_reg_count;
   a_reg_count = 0;
	//a = 0;
   //a+=1;
    int th_id = blockIdx.x * blockDim.x + threadIdx.x;

	a_reg_count+=1;
	//if(index < 256){
	a[th_id] = a_reg_count;
	//}
	//int reg_count = 0;
//	reg_count+=1;
	//*count+=1;
	//*count += a[th_id];

	//    __syncthreads();

	while( n > 0 )
	{
		n = (n / 2);
		if( th_id < n )
		{
			a[ th_id ] += a[ th_id + n ];
		}
	}
	// Observe that there is no need for synchronisation
	// here because n<32 and all threads in a warp are
	// "step-locked"; i.e., they must walk through the
	// instructions one-by-one together at the same time 
	
	// We need to ensure that this is executed by a thread
	// on the first warp; otherwise, a thread on a different
	// warp could perform this write before the value is ready. 
	if( th_id == 0 )
	{
		*count = a[ 0 ];
	}

	//atomicAdd(count, 1);
    
}

__global__
void abcd(int *count, int n, int n_phase_2) {
   __shared__ int a[1024];

    int th_id = blockIdx.x * blockDim.x + threadIdx.x;

int h[1000];
	int a_reg_count = 1;
//phase 1: n/t = 100

/* 
a[th_id] = 0;
for(int x = 0; x < 100; x++){

	a[th_id] += 1;
	
	
}
if(th_id == 999){
			printf("%d\n", a[th_id]);

			}

			*/

	for( int i = ( th_id * (100) ); i < ( (th_id + 1) * (100) ); i++){

		//a[i] = a_reg_count;
		a[th_id] +=1;
		if(th_id == 999)
			printf("%d, %d\n", a[th_id], i);

	}


__syncthreads();

//phase 2:


	while( n_phase_2 > 0 )
	{
		n_phase_2 = (n_phase_2 / 2);
		if( th_id < n_phase_2 )
		{
			a[ th_id ] += a[ th_id + n_phase_2 ];
				//		printf("%d\n\n",n_phase_2);

		}
	}
	// Observe that there is no need for synchronisation
	// here because n<32 and all threads in a warp are
	// "step-locked"; i.e., they must walk through the
	// instructions one-by-one together at the same time 
	
	// We need to ensure that this is executed by a thread
	// on the first warp; otherwise, a thread on a different
	// warp could perform this write before the value is ready. 
	if( th_id == 0 )
	{
		*count = a[ 0 ];
	}

	//atomicAdd(count, 1);

//	*count = (th_id) * (n/1000);
    
}

int main() {
	int h_count[1];
	int *d_count;

	auto n = 100000;
	auto n_phase_2 = 1024;
	auto d_n = 256;
	h_count[0] = 0;
	hipMalloc((void**)&d_count, sizeof(int));
	hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

	abcd << <1, 1000 >> > (d_count, n, n_phase_2);


	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
   // hipDeviceSynchronize();



	std::cout << h_count[0] <<std::endl;
}